/*********************************************************************
* Filename:   arcfour_test.c
* Author:     Brad Conte (brad AT bradconte.com)
* Copyright:
* Disclaimer: This code is presented "as is" without any guarantees.
* Details:    Performs known-answer tests on the corresponding ARCFOUR
              implementation. These tests do not encompass the full
              range of available test vectors, however, if the tests
              pass it is very, very likely that the code is correct
              and was compiled properly. This code also serves as
              example usage of the functions.
*********************************************************************/

/*************************** HEADER FILES ***************************/
#include <stdio.h>
#include <memory.h>
#include "arcfour.h"

/*********************** FUNCTION DEFINITIONS ***********************/
int rc4_test()
{
    BYTE state[256];
    BYTE key[3][10] = {{"Key"}, {"Wiki"}, {"Secret"}};
    BYTE stream[3][10] = {{0xEB,0x9F,0x77,0x81,0xB7,0x34,0xCA,0x72,0xA7,0x19},
                          {0x60,0x44,0xdb,0x6d,0x41,0xb7},
                          {0x04,0xd4,0x6b,0x05,0x3c,0xa8,0x7b,0x59}};
    int stream_len[3] = {10,6,8};
    BYTE buf[1024];
    int idx;
    int pass = 1;

    BYTE *d_state;
    BYTE *d_key;
    BYTE *d_stream;
    BYTE *d_buf;

    hipMalloc(&d_state,  sizeof(BYTE) * 256);
    hipMalloc(&d_key,    sizeof(BYTE) * 3 * 10);
    hipMalloc(&d_stream, sizeof(BYTE) * 3 * 10);
    hipMalloc(&d_buf,    sizeof(BYTE) * 1024);
    hipMalloc(&d_stream_len, sizeof(BYTE) * 3);

    hipMemcpy(d_state, state,           sizeof(BYTE) * 256,    hipMemcpyHostToDevice);
    hipMemcpy(d_key, key,               sizeof(BYTE) * 3 * 10, hipMemcpyHostToDevice);
    hipMemcpy(d_stream, stream,         sizeof(BYTE) * 3 * 10, hipMemcpyHostToDevice);
    hipMemcpy(d_buf, buf,               sizeof(BYTE) * 1024,   hipMemcpyHostToDevice);
    hipMemcpy(d_stream_len, stream_len, sizeof(BYTE) * 3,      hipMemcpyHostToDevice);

    // Only test the output stream. Note that the state can be reused.
    for (idx = 0; idx < 3; idx++) {
        arcfour_key_setup<<<4,256>>>(d_state, d_key[idx], d_strlen(d_key[idx]));
        arcfour_generate_stream(d_state,d_ buf, d_stream_len[idx]);

        hipMemcpy(stream, d_stream,         sizeof(BYTE) * 3 * 10, hipMemcpyDeviceToHost);
        hipMemcpy(buf, d_buf,               sizeof(BYTE) * 1024,   hipMemcpyDeviceToHost);
        hipMemcpy(stream_len, d_stream_len, sizeof(BYTE) * 3,      hipMemcpyDeviceToHost);

        pass = pass && !memcmp(stream[idx], buf, stream_len[idx]);
    }


    hipFree(d_state);
    hipFree(d_key);
    hipFree(d_stream);
    hipFree(d_buf);
    hipFree(d_stream_len);

    return(pass);
}

int main()
{
    printf("ARCFOUR tests: %s\n", rc4_test() ? "SUCCEEDED" : "FAILED");

    return(0);
}
