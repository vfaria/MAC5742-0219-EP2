/*********************************************************************
* Filename:   rot-13_test.c
* Author:     Brad Conte (brad AT bradconte.com)
* Copyright:
* Disclaimer: This code is presented "as is" without any guarantees.
* Details:    Performs known-answer tests on the corresponding ROT-13
              implementation. These tests do not encompass the full
              range of available test vectors, however, if the tests
              pass it is very, very likely that the code is correct
              and was compiled properly. This code also serves as
              example usage of the functions.
*********************************************************************/

/*************************** HEADER FILES ***************************/
#include <stdio.h>
#include <string.h>
#include "rot-13cu.h"

/*********************** FUNCTION DEFINITIONS ***********************/
int rot13_test()
{
    char text[] = {"ABCDEFGHIJKLMNOPQRSTUVWXYZabcdefghijklmnopqrstuvwxyz"};
    char code[] = {"NOPQRSTUVWXYZABCDEFGHIJKLMnopqrstuvwxyzabcdefghijklm"};
    char buf[1024];
    int pass = 1;
    int len;

    char* d_buf;
    int* d_len;

    if(hipMalloc(&d_buf, sizeof(char) * 1024) != hipSuccess)
    {
        return 0;
    }

    if(hipMalloc(&d_len, sizeof(int)) != hipSuccess)
    {
        return 0;
    }

    strcpy(buf, text);
    len = strlen(text);

    if(hipMemcpy(d_buf, buf, sizeof(char) * 1024, hipMemcpyHostToDevice) != hipSuccess)
    {
        hipFree(d_buf);
        hipFree(d_len);
        return 0;
    }

    if(hipMemcpy(d_len, &len, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
    {
        hipFree(d_buf);
        hipFree(d_len);
        return 0;
    }

    // To encode, just apply ROT-13.
    rot13<<<4, 256>>>(d_buf, len);

    if(hipMemcpy(buf, d_buf, sizeof(char) * 1024, hipMemcpyDeviceToHost) != hipSuccess)
    {
        hipFree(d_buf);
        return 0;
    }
    
    pass = pass && !strcmp(code, buf);

    // To decode, just re-apply ROT-13.
    rot13<<<4, 256>>>(d_buf, len);

    if(hipMemcpy(buf, d_buf, sizeof(char) * 1024, hipMemcpyDeviceToHost) != hipSuccess)
    {
        hipFree(d_buf);
        return 0;
    }

    pass = pass && !strcmp(text, buf);

    hipFree(d_buf);
    hipFree(d_len);

    return(pass);
}

int main()
{
    printf("ROT-13 tests with CUDA: %s\n", rot13_test() ? "SUCCEEDED" : "FAILED");

    return(0);
}
