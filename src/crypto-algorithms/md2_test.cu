/*********************************************************************
* Filename:   md2_test.c
* Author:     Brad Conte (brad AT bradconte.com)
* Copyright:
* Disclaimer: This code is presented "as is" without any guarantees.
* Details:    Performs known-answer tests on the corresponding MD2
              implementation. These tests do not encompass the full
              range of available test vectors, however, if the tests
              pass it is very, very likely that the code is correct
              and was compiled properly. This code also serves as
              example usage of the functions.
*********************************************************************/

/*************************** HEADER FILES ***************************/
#include <stdio.h>
#include <string.h>
#include <memory.h>
#include "md2cu.h"

/*********************** FUNCTION DEFINITIONS ***********************/
int md2_test()
{
    BYTE text1[] = {"abc"};
    BYTE text2[] = {"abcdefghijklmnopqrstuvwxyz"};
    BYTE text3_1[] = {"ABCDEFGHIJKLMNOPQRSTUVWXYZabcde"};
    BYTE text3_2[] = {"fghijklmnopqrstuvwxyz0123456789"};

    BYTE *d_text;

    BYTE hash1[MD2_BLOCK_SIZE] = {0xda,0x85,0x3b,0x0d,0x3f,0x88,0xd9,0x9b,0x30,0x28,0x3a,0x69,0xe6,0xde,0xd6,0xbb};
    BYTE hash2[MD2_BLOCK_SIZE] = {0x4e,0x8d,0xdf,0xf3,0x65,0x02,0x92,0xab,0x5a,0x41,0x08,0xc3,0xaa,0x47,0x94,0x0b};
    BYTE hash3[MD2_BLOCK_SIZE] = {0xda,0x33,0xde,0xf2,0xa4,0x2d,0xf1,0x39,0x75,0x35,0x28,0x46,0xc3,0x03,0x38,0xcd};

    BYTE buf[16];
    BYTE *d_buf;

    BYTE *d_ctx;
    int *d_ctx_len;  
    
    int pass = 1;

    // Device structures used in all tests:
    hipMalloc(&d_ctx, 80 * sizeof(BYTE));
    hipMalloc(&d_ctx_len, sizeof(int));
    hipMalloc(&d_buf, 16 * sizeof(BYTE));

    // Test for text1
    hipMalloc(&d_text, strlen((char *) text1) * sizeof(BYTE));
    hipMemcpy(d_text, &text1, strlen((char *) text1) * sizeof(BYTE), hipMemcpyHostToDevice);

    md2_init<<<1, 1>>>(d_ctx, d_ctx_len);
    md2_update<<<1, 1>>>(d_ctx, d_ctx_len, d_text, strlen((char *) text1));
    md2_final<<<1, 1>>>(d_ctx, d_ctx_len, d_buf);

    hipMemcpy(&buf, d_buf, 16 * sizeof(BYTE), hipMemcpyDeviceToHost);
    hipFree(d_text);

    pass = pass && !memcmp(hash1, buf, MD2_BLOCK_SIZE);

    // Test for text2
    hipMalloc(&d_text, strlen((char *) text2) * sizeof(BYTE));
    hipMemcpy(d_text, &text2, strlen((char *) text2) * sizeof(BYTE), hipMemcpyHostToDevice);

    md2_init<<<1, 1>>>(d_ctx, d_ctx_len);
    md2_update<<<1, 1>>>(d_ctx, d_ctx_len, d_text, strlen((char *) text2));
    md2_final<<<1, 1>>>(d_ctx, d_ctx_len, d_buf);

    hipMemcpy(&buf, d_buf, 16 * sizeof(BYTE), hipMemcpyDeviceToHost);
    hipFree(d_text);

    pass = pass && !memcmp(hash2, buf, MD2_BLOCK_SIZE);

    // Test for text3
    hipMalloc(&d_text, strlen((char *) text3_1) * sizeof(BYTE));
    hipMemcpy(d_text, &text3_1, strlen((char *) text3_1) * sizeof(BYTE), hipMemcpyHostToDevice);

    md2_init<<<1, 1>>>(d_ctx, d_ctx_len);
    md2_update<<<1, 1>>>(d_ctx, d_ctx_len, d_text, strlen((char *) text3_1));
    hipFree(d_text);

    hipMalloc(&d_text, strlen((char *) text3_2) * sizeof(BYTE));
    hipMemcpy(d_text, &text3_2, strlen((char *) text3_2) * sizeof(BYTE), hipMemcpyHostToDevice);
    md2_update<<<1, 1>>>(d_ctx, d_ctx_len, d_text, strlen((char *) text3_2));
    md2_final<<<1, 1>>>(d_ctx, d_ctx_len, d_buf);

    hipMemcpy(&buf, d_buf, 16 * sizeof(BYTE), hipMemcpyDeviceToHost);
    hipFree(d_text);

    pass = pass && !memcmp(hash3, buf, MD2_BLOCK_SIZE);

    hipFree(d_ctx);
    hipFree(d_ctx_len);

    return(pass);
}

int main()
{
    printf("MD2 tests: %s\n", md2_test() ? "SUCCEEDED" : "FAILED");
}
